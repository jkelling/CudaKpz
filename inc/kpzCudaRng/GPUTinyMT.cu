#include "hip/hip_runtime.h"
/***************************************************************************
*   Copyright 2014 - 2016 Jeffrey Kelling <j.kelling@hzdr.de>
*                  Helmholtz-Zentrum Dresden-Rossendorf
*                  Institute of Ion Beam Physics and Materials Research
*
*	This file is part of CudaKpz.
*
*   CudaKPZ is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   CudaKPZ is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with CudaKPZ.  If not, see <http://www.gnu.org/licenses/>.
***************************************************************************/

#include "GPUTinyMT.h"
#include "GPUTinyMT.cuh"

#include "../tinyMT/tinymt32_param.h"

#include <cassert>
#include <climits>
#include <iostream>
#include <sstream>
#include <string>
#include <memory>
#include <stdexcept>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipError_t.h>

#include <H5Cpp.h>

void TinyMT32::initCUDA()
{
	{ // round up to multiples or warps
		int tmp = 0;
		hipDeviceProp_t prop;
		CUDA_SAFE_CALL_THROW(hipGetDevice(&tmp));
		CUDA_SAFE_CALL_THROW(hipGetDeviceProperties(&prop, tmp));
		if(m_generatorCount%prop.warpSize)
			m_generatorCount = (m_generatorCount/prop.warpSize+1)*prop.warpSize;
	}

	CUDA_SAFE_CALL(hipMalloc((void**)&d_Random, m_generatorCount*sizeof(unsigned int)*7));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_tmat, m_generatorCount*sizeof(unsigned int)));
	if(!d_Random || !d_tmat)
	{
		throw std::runtime_error("Insufficient devicememory for TinyMT state.");
	}
	CUDA_SAFE_CALL_THROW( hipMemcpy(d_Random+4*m_generatorCount, \
				tinyMTmat1, m_generatorCount*sizeof(unsigned int), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL_THROW( hipMemcpy(d_Random+5*m_generatorCount, \
				tinyMTmat2, m_generatorCount*sizeof(unsigned int), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL_THROW( hipMemcpy(d_Random+6*m_generatorCount, \
				tinyMTtmat, m_generatorCount*sizeof(unsigned int), hipMemcpyHostToDevice) );
	if(KMC_N_TMT_PARAM_SETS < m_generatorCount)
	{
		std::ostringstream os;
		os << "Too many TinyMT parametersets requested: " << m_generatorCount << " > " << KMC_N_TMT_PARAM_SETS << ".";
		throw std::runtime_error(os.str());
	}
	std::cout << "TinyMT32: Using parametersets: 0 .. " << m_generatorCount-1 << '\n';

	int device;
	CUDA_SAFE_CALL(hipGetDevice(&device));
	hipDeviceProp_t prop;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, device));
	m_blocks = prop.multiProcessorCount;
	m_threads = m_generatorCount/m_blocks;
	if(m_generatorCount%m_blocks)
		m_blocks += 1;
}

__global__ void seedGPU(unsigned int* d_Random, size_t generatorCount)
{
	if(blockIdx.x*blockDim.x+threadIdx.x < generatorCount)
	{
		TinyMT32::Device rng(d_Random);
			rng.seed();
	}
	__syncthreads();
}

void TinyMT32::randomize()
{
	assert(m_dsfmt != 0);
	std::vector<unsigned int> rnd(m_generatorCount);
	for (int a = 0; a<m_generatorCount; ++a)
	{
		rnd[a] = dsfmt_genrand_close_open(m_dsfmt)*UINT_MAX;
	}
	randomize(rnd);
}

void TinyMT32::randomize(const std::vector<unsigned int>& seeds)
{
	assert(m_generatorCount <= KMC_N_TMT_PARAM_SETS);
	CUDA_SAFE_CALL_THROW( hipMemcpy(d_Random, seeds.data(), m_generatorCount*sizeof(unsigned int), hipMemcpyHostToDevice) );

	// determine optimal <<<blocks,threads>>> for seeding
	hipDeviceProp_t prop;
	int tmp;
	CUDA_SAFE_CALL_THROW(hipGetDevice(&tmp));
	CUDA_SAFE_CALL_THROW(hipGetDeviceProperties(&prop, tmp));
	const int maxThreads = prop.maxThreadsPerBlock/prop.warpSize;
	const int maxBlocks = m_generatorCount/prop.warpSize;
	const int minBlocks = prop.multiProcessorCount;
	int blocks, threads;
	if(maxBlocks < minBlocks || maxBlocks % minBlocks)
	{ // the latter case seems very unlikely to occour in any practical application
		blocks = maxBlocks;
		threads = prop.warpSize;
	}
	else
	{
		blocks = maxBlocks/minBlocks;
		threads = m_generatorCount/blocks;
		while(threads > maxThreads)
		{
			threads >>= 1;
			blocks <<= 1;
		}
	}

	std::cerr << "running " << blocks << ' ' << threads << " to seed tmt\n";
	seedGPU<<<blocks, threads>>> (d_Random, m_generatorCount);
	CUDA_SAFE_CALL_THROW( hipGetLastError() );
}

bool TinyMT32::minGenerators(unsigned int ngenmin)
{
	if(m_generatorCount >= ngenmin)
		return true;

	if (d_Random)
		CUDA_SAFE_CALL(hipFree(d_Random));
	m_generatorCount = ngenmin;
	initCUDA();
	return false;
}

TinyMT32::~TinyMT32()
{
	if (d_Random)
		CUDA_SAFE_CALL(hipFree(d_Random));
}

#ifdef USE_LIB_SPLASH
#include <KMCsplash.h>

void TinyMT32::writeH5(splash::DataCollector* data, int id, const char* prefix)
{
	std::string name;
	if(prefix)
		name = prefix;
	name += ".tinyMT";

	auto rnd = std::unique_ptr<unsigned int[]>( new unsigned int[m_generatorCount*7] );
	CUDA_SAFE_CALL_THROW( hipMemcpy(rnd.get(), d_Random, m_generatorCount*sizeof(unsigned int)*7, hipMemcpyDeviceToHost) );
	data->write(id, KmcSplash::ColTypeUInt32, 2, splash::Selection(splash::Dimensions(m_generatorCount, 7, 1)), name.c_str(), rnd.get());
}

bool TinyMT32::readH5(splash::DataCollector* data, int id, const char* prefix)
{
	std::string name;
	if(prefix)
		name = prefix;
	name += ".tinyMT";

	splash::Dimensions size;
	try {
		data->read(id, name.c_str(), size, 0);
	}
	catch (splash::DCException) {
		return false;
	}

	if(size[0] != m_generatorCount)
	{
		std::cerr << "[GPUTinyMT][readH5] incompatible tinyMT state in HDF5 file.\n";
		return false;
	}
	auto rnd = std::unique_ptr<unsigned int[]>( new unsigned int[m_generatorCount*7] );
	data->read(id, name.c_str(), size, rnd.get());
	CUDA_SAFE_CALL_THROW( hipMemcpy(d_Random, rnd.get(), m_generatorCount*sizeof(unsigned int)*7, hipMemcpyHostToDevice) );
	std::cout << "[GPUTinyMT][readH5] restored TinyMT state from file.\n";
	return true;
}
#endif

#include <H5Cpp.h>

static std::string mkH5DsName(const char* prefix)
{
	std::string name;
	if(prefix)
	{
		name = prefix;
		name += ".tinyMT";
	}
	else
		name = "tinyMT";

	return name;
}

void TinyMT32::writeH5(H5::CommonFG& dest, const char* prefix)
{
	const auto name = ::mkH5DsName(prefix);

	const hsize_t size[2] = {m_generatorCount, 7};
	auto rnd = std::unique_ptr<unsigned int[]>( new unsigned int[size[0]*size[1]] );
	CUDA_SAFE_CALL_THROW( hipMemcpy(rnd.get(), d_Random, size[0]*size[1]*sizeof(unsigned int), hipMemcpyDeviceToHost) );

	H5::DataSpace dspace(2, size);
	auto dset = dest.createDataSet(name, H5::PredType::STD_U32LE, dspace);
	dset.write(rnd.get(), H5::PredType::STD_U32LE);
}

void TinyMT32::readH5(H5::CommonFG& src, const char* prefix)
{
	const auto name = ::mkH5DsName(prefix);

	auto dset = src.openDataSet(name);
	auto dspace = dset.getSpace();
	if(dspace.getSimpleExtentNdims() != 2 )
		throw std::domain_error("[TinyMT32::readH5] Invalid tinyMT DataSet: Wrong NDim");
	hsize_t size[2];
	dspace.getSimpleExtentDims(size);
	if(size[1] != 7)
		throw std::domain_error("[TinyMT32::readH5] Invalid tinyMT DataSet: Wrong number of sections ([1] != 7).");
	if(size[0] != m_generatorCount)
		throw std::range_error("[TinyMT32::readH5] Incompatible tinyMT state in HDF5 file.\n");

	auto rnd = std::unique_ptr<unsigned int[]>( new unsigned int[m_generatorCount*7] );
	dset.read(rnd.get(), H5::PredType::STD_U32LE);
	CUDA_SAFE_CALL_THROW( hipMemcpy(d_Random, rnd.get(), m_generatorCount*sizeof(unsigned int)*7, hipMemcpyHostToDevice) );

	std::cout << "[GPUTinyMT][readH5] restored TinyMT state from file.\n";
}
