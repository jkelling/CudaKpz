/***************************************************************************
*   Copyright 2014 - 2014 Jeffrey Kelling <j.kelling@hzdr.de>
*                  Helmholtz-Zentrum Dresden-Rossendorf
*                  Institute of Ion Beam Physics and Materials Research
*
*	This file is part of CudaKpz.
*
*   CudaKPZ is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   CudaKPZ is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with CudaKPZ.  If not, see <http://www.gnu.org/licenses/>.
***************************************************************************/

#include "GPUSLCG64.h"

#include "../kmcRandom.h"

#include <cassert>

#include <hip/hip_runtime.h>
#include <hipError_t.h>

// dummy declarations, static SLCG64::Device::random() is not be used here
/*unsigned long long KMC_SLCG_A_SKIP = 0;*/
/*unsigned long long KMC_SLCG_C_SKIP = 0;*/

void SLCG64::initCUDA()
{
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Random, m_generatorCount*sizeof(unsigned long long)));
	if(!d_Random)
	{
		std::cerr << "Insufficient devicememory for random seeds.\n";
		exit(0);
	}
}

void SLCG64::randomize()
{
	unsigned long long* rnd = new unsigned long long[m_generatorCount];
	rnd[0] = (unsigned long long)(dsfmt_genrand_close_open(m_dsfmt)*((unsigned)-1))
		^ (((unsigned long long)(dsfmt_genrand_close_open(m_dsfmt)*((unsigned)-1)))<<32);
	for (int a = 1; a<m_generatorCount; ++a)
	{
		rnd[a] = SLCGen(rnd[a-1]);
	}
	CUDA_SAFE_CALL( hipMemcpy(d_Random, rnd, m_generatorCount*sizeof(unsigned long long), hipMemcpyHostToDevice) );
	delete[] rnd;
}

SLCG64::~SLCG64()
{
	if (d_Random)
		CUDA_SAFE_CALL(hipFree(d_Random));
}
